
#include <hip/hip_runtime.h>


__global__ void VectorAdd_Kernel(const double *a, const double *b, double *c, const int n) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

void VectorAdd_GPU(const double *h_a, const double *h_b, double *h_c, const int n) {

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Number of threads in each thread block
    int blockSize = 10000;

    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    VectorAdd_Kernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}