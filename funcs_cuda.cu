#include "hip/hip_runtime.h"

#include "funcs_cuda.cuh"
// Number of threads in each thread block
const int blockSize = 16384;


__global__ void VectorAdd_Kernel(const double *a, const double *b, double *c, const int n) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

void VectorAdd_GPU(const double *h_a, const double *h_b, double *h_c, const int n) {

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
    // Device input vectors
    double *dd_a;
    double *dd_b;
    //Device output vector
    double *dd_c;
    // Allocate memory for each vector on GPU
    hipMalloc(&dd_a, bytes);
    hipMalloc(&dd_b, bytes);
    hipMalloc(&dd_c, bytes);
    // Copy host vectors to device
    hipMemcpy( dd_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( dd_b, h_b, bytes, hipMemcpyHostToDevice);

    // Number of threads in each thread block
//    int blockSize = 10000;

    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    VectorAdd_Kernel<<<gridSize, blockSize>>>(dd_a, dd_b, dd_c, n);
    hipMemcpy( h_c, dd_c, bytes, hipMemcpyDeviceToHost );
    // Release device memory
    hipFree(dd_a);
    hipFree(dd_b);
    hipFree(dd_c);
}

void VectorsClass::VectorAdd_GPU_InClass(const double *h_a, const double *h_b, double *h_c, const int n) {
    // Copy host vectors to device
    hipMemcpy( this->d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( this->d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Number of threads in each thread block
//    int blockSize = 10000;

    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    VectorAdd_Kernel<<<gridSize, blockSize>>>(this->d_a, this->d_b, this->d_c, n);
    hipMemcpy( h_c, this->d_c, bytes, hipMemcpyDeviceToHost );
    // Release device memory

}
void VectorsClass::VectorAdd_GPU_InClass(double *h_c, const int n) {

    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    VectorAdd_Kernel<<<gridSize, blockSize>>>(this->d_a, this->d_b, this->d_c, n);
    hipMemcpy( h_c, this->d_c, bytes, hipMemcpyDeviceToHost );
    // Release device memory

}