#include "hip/hip_runtime.h"

#include "funcs_cuda.cuh"
// Number of threads in each thread block
const int blockSize = 16384;


__global__ void VectorAdd_Kernel(const double *a, const double *b, double *c, const int n) {
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

void VectorAdd_GPU(const double *h_a, const double *h_b, double *h_c, const int n) {

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
    // Device input vectors
    double *dd_a;
    double *dd_b;
    //Device output vector
    double *dd_c;
    // Allocate memory for each vector on GPU
    hipMalloc(&dd_a, bytes);
    hipMalloc(&dd_b, bytes);
    hipMalloc(&dd_c, bytes);
    // Copy host vectors to device
    hipMemcpy( dd_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( dd_b, h_b, bytes, hipMemcpyHostToDevice);

    // Number of threads in each thread block
//    int blockSize = 10000;

    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    VectorAdd_Kernel<<<gridSize, blockSize>>>(dd_a, dd_b, dd_c, n);
    hipMemcpy( h_c, dd_c, bytes, hipMemcpyDeviceToHost );
    // Release device memory
    hipFree(dd_a);
    hipFree(dd_b);
    hipFree(dd_c);
}

void VectorsClass::VectorAdd_GPU_InClass(const double *h_a, const double *h_b, double *h_c, const int n) {
    // Copy host vectors to device
    hipMemcpy( this->d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( this->d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Number of threads in each thread block
//    int blockSize = 10000;

    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    VectorAdd_Kernel<<<gridSize, blockSize>>>(this->d_a, this->d_b, this->d_c, n);
    hipMemcpy( h_c, this->d_c, bytes, hipMemcpyDeviceToHost );
    // Release device memory

}
void VectorsClass::VectorAdd_GPU_InClass(double *h_c, const int n) {

    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    VectorAdd_Kernel<<<gridSize, blockSize>>>(this->d_a, this->d_b, this->d_c, n);
    hipMemcpy( h_c, this->d_c, bytes, hipMemcpyDeviceToHost );
    // Release device memory

}

void MatricesClass::initialize_A_B(const double *h_A, const double *h_B, const int m, const int n, const int k) {
    checkCudaErrors(hipMemcpy( this->d_A, h_A, m*n*sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( this->d_B, h_B, n*k*sizeof(double), hipMemcpyHostToDevice));

}

void MatricesClass::matmul_GPU() {
    checkCudaErrors(hipblasCreate(&this->handle));
    // Calculate: c = (alpha*a) * b + (beta*c)
    // MxN = MxK * KxN
    // Signature: handle, operation, operation, M, N, K, alpha, A, lda, B, ldb,
    // beta, C, ldc
    double alpha {1.0};
    double beta {0.0};
    checkCudaErrors(hipblasDgemm(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, this->m, this->k, this->n, &alpha, this->d_A, this->m, this->d_B, this->n,
                    &beta, this->d_C, this->m));
    hipDeviceSynchronize();
}
void MatricesClass::retrieve_C(double *h_C) {
//    std::cout << "inside retrive_C" << std::endl;
    checkCudaErrors(hipMemcpy( h_C, this->d_C, this->bytes_C, hipMemcpyDeviceToHost ));
//    for (int i {0}; i < m*k; ++i) {
//        std::cout << h_C[i] << std::endl;
//    }
}
void MatricesClassEigen::initialize_A_B(const Eigen::MatrixXd &A, const Eigen::MatrixXd &B) {
    checkCudaErrors(hipMemcpy( this->d_A, A.data(), A.rows()*A.cols()*sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( this->d_B, B.data(), B.rows()*B.cols()*sizeof(double), hipMemcpyHostToDevice));

}

void MatricesClassEigen::matmul_GPU() {
    checkCudaErrors(hipblasCreate(&this->handle));
    // Calculate: c = (alpha*a) * b + (beta*c)
    // MxN = MxK * KxN
    // Signature: handle, operation, operation, M, N, K, alpha, A, lda, B, ldb,
    // beta, C, ldc
    double alpha {1.0};
    double beta {0.0};
    checkCudaErrors(hipblasDgemm(this->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, this->m, this->k, this->n, &alpha, this->d_A, this->m, this->d_B, this->n,
                                &beta, this->d_C, this->m));
    hipDeviceSynchronize();
}
void MatricesClassEigen::retrieve_C(Eigen::MatrixXd &C) {
//    std::cout << "inside retrive_C" << std::endl;
    checkCudaErrors(hipMemcpy( C.data(), this->d_C, this->bytes_C, hipMemcpyDeviceToHost ));
//    for (int i {0}; i < m*k; ++i) {
//        std::cout << h_C[i] << std::endl;
//    }
}